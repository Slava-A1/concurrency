#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include "kmeans.h"
#include <iostream>

#define CHECK_CUDA1(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        } \
    } while (0)

//Credit to Rishi and Oakridge National Labs (which is where I think GPT pulled his macro from)
#define CHECK_CUDA(call) ({ \
    hipError_t err = call; \
    if(err != hipSuccess){ \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
    } \
})

__global__ void addVectors(float* a, float* b, float* res, int sz){
    int myIndex = (blockIdx.x * blockDim.x) + threadIdx.x;

    if(myIndex < sz){
        res[myIndex] = a[myIndex] + b[myIndex];
    }
} 

void cudaBasicTest(const std::vector<float>& a, const std::vector<float>& b, std::vector<float>& out){
    const int sz = a.size();
    float *daPtr, *dbPtr, *dresPtr;
    size_t allocSize = sizeof(float) * sz;
    CHECK_CUDA(hipMalloc(&daPtr, allocSize));
    CHECK_CUDA(hipMalloc(&dbPtr, allocSize));
    CHECK_CUDA(hipMalloc(&dresPtr, allocSize));
     
    CHECK_CUDA(hipMemcpy(daPtr, a.data(), allocSize, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dbPtr, b.data(), allocSize, hipMemcpyHostToDevice));

    int threadsPerBlock = 256; //why not
    int nBlocks = (int)std::ceil(((float)sz) / threadsPerBlock);

    addVectors<<<nBlocks, threadsPerBlock>>>(daPtr, dbPtr, dresPtr, sz);
    hipError_t lastErr = hipGetLastError();
    if(lastErr != hipSuccess){
        std::cerr << "CUDA Error: " << hipGetErrorString(lastErr) << std::endl;
    }
    
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipMemcpy(out.data(), dresPtr, allocSize, hipMemcpyDeviceToHost));

    CHECK_CUDA(hipFree(daPtr));
    CHECK_CUDA(hipFree(dbPtr));
    CHECK_CUDA(hipFree(dresPtr));
}

//number of get distance ops is num_points * num_clusters
//Idea: one block for each get distance, with one thread per distance operation (a - b)^2

//Idea: use critical section to update max in parallel?
//Better: store distances in array, then have another kernel find max for each point
//Technically a parallel reduction would be best but not sure if there will be enough time to write that

/*
//Returns the centroids and populates the passed in vector of integers with the centroid id for each point
std::vector<std::vector<float>> genCentroidPar(const std::vector<std::vector<float>>& data, std::vector<int> centroidIds){
    //Set up
    const int nPoints = data.size();
    std::vector<std::vector<float>> centroids = getRandomCentroids(data);
    //Stores number of points associated with each centroid
    std::vector<int> numAssocPoints(num_cluster);
    int currIter = 1;
    bool done = false;
    float currConv = 1.0;

    //Making everything accessible to CUDA
    float** centroidsPtr = new float*[num_cluster];
    for(int i = 0; i < num_cluster; ++i){
        //extracting pointers from the vectors
        centroidsPtr[i] = centroids[i].data();
    }

    while(!done){
        float conv = 0.0;
        
        std::fill(numAssocPoints.begin(), numAssocPoints.end(), 0);
        //Stores sums of values for each centroid, will be divided later
        std::vector<std::vector<float>> nextCentroids(num_cluster, std::vector<float>(dims, 0.0));
        
        //Finding closest centroid to each point
        for(int i = 0; i < nPoints; ++i){
            std::pair<int, float> closePair = findClosestCentroid(data[i], centroids);
            centroidIds[i] = closePair.first;
            conv += closePair.second;

            //One more point associated with the selected centroid
            numAssocPoints[closePair.first] += 1;
            //add in values for later division
            for(int j = 0; j < dims; ++j){
                nextCentroids[closePair.first][j] += data[i][j];
            }
        }
        
        //updating centroids
        for(int i = 0; i < num_cluster; ++i){
            updateCentroid(nextCentroids[i], numAssocPoints[i]);
        }
        
        centroids = std::move(nextCentroids);
        
        float improvementAmt = std::abs(currConv - conv) / currConv;
        currConv = conv;

        done = (currIter == max_num_iter) || (improvementAmt < threshold);
        ++currIter;
    }
    
    delete centroidsPtr;

    //Returns final centroids in the case that they need to be printed
    return centroids;
}

*/
