#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include "kmeans.h"

//number of get distance ops is num_points * num_clusters
//Idea: one block for each get distance, with one thread per distance operation (a - b)^2

//Idea: use critical section to update max in parallel?
//Better: store distances in array, then have another kernel find max for each point
//Technically a parallel reduction would be best but not sure if there will be enough time to write that

//Returns the centroids and populates the passed in vector of integers with the centroid id for each point
std::vector<std::vector<float>> genCentroidPar(const std::vector<std::vector<float>>& data, std::vector<int> centroidIds){
    //Set up
    const int nPoints = data.size();
    std::vector<std::vector<float>> centroids = getRandomCentroids(data);
    //Stores number of points associated with each centroid
    std::vector<int> numAssocPoints(num_cluster);
    int currIter = 1;
    bool done = false;
    float currConv = 1.0;

    //Making everything accessible to CUDA
    float** centroidsPtr = new float*[num_cluster];
    for(int i = 0; i < num_cluster; ++i){
        //extracting pointers from the vectors
        centroidsPtr[i] = centroids[i].data();
    }

    while(!done){
        float conv = 0.0;
        
        std::fill(numAssocPoints.begin(), numAssocPoints.end(), 0);
        //Stores sums of values for each centroid, will be divided later
        std::vector<std::vector<float>> nextCentroids(num_cluster, std::vector<float>(dims, 0.0));
        
        //Finding closest centroid to each point
        for(int i = 0; i < nPoints; ++i){
            std::pair<int, float> closePair = findClosestCentroid(data[i], centroids);
            centroidIds[i] = closePair.first;
            conv += closePair.second;

            //One more point associated with the selected centroid
            numAssocPoints[closePair.first] += 1;
            //add in values for later division
            for(int j = 0; j < dims; ++j){
                nextCentroids[closePair.first][j] += data[i][j];
            }
        }
        
        //updating centroids
        for(int i = 0; i < num_cluster; ++i){
            updateCentroid(nextCentroids[i], numAssocPoints[i]);
        }
        
        centroids = std::move(nextCentroids);
        
        float improvementAmt = std::abs(currConv - conv) / currConv;
        currConv = conv;

        done = (currIter == max_num_iter) || (improvementAmt < threshold);
        ++currIter;
    }
    
    delete centroidsPtr;

    //Returns final centroids in the case that they need to be printed
    return centroids;
}
